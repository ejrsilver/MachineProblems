#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <random>
#include "hip/hip_runtime.h"
#include ""

/*
Problem 2: Matrix Addition

Steps: - DONE
1. Allocate input and output in host code
2. kernel with one output matrix element per thread (16x16 thread blocks)
3. kernel with one output matrix row (16 threads per block)
4. kernel with one output matrix column (16 threads per block)

Analyse pros and cons of above approaches

Create Random matrices A and B:
- 125x125
- 250x250
- 500x500
- 1000x1000
- 2000x2000

Analytics:
- kernel execution time
- execute multiple times and report averages/remove outliers
- compare CPU and GPU performance using graphs/tables

CUDA Events: - DONE
- start time
- stop time
- begin recording
- stop recording
- find elapsed time
- destroy events
*/

#define B_WIDTH 16

__global__ void matrixAdd(float* A, float* B, float* C, int n) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n) {
		int i = y*n + x;
		C[i] = A[i] + B[i];
	}
	__syncthreads();
}

__global__ void matrixAddRow(float* A, float* B, float* C, int n) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < n) {
		for (int y = 0; y < n; y++) {
			int i = y*n + x;
			C[i] = A[i] + B[i];
		}
	}
	__syncthreads();
}

__global__ void matrixAddColumn(float* A, float* B, float* C, int n) {
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < n) {
		for (int x = 0; x < n; x++) {
			int i = y*n + x;
			C[i] = A[i] + B[i];
		}
	}
	__syncthreads();
}

void cudaAdd(const float* A, const float* B, float* C, int n, int mode) {
	int m_size = n*n;
	// allocate gpu pointers and check for errors
	float* dev_A;
	float* dev_B;
	float* dev_C;

	hipError_t gpu_error = hipMalloc((void**)&dev_A, m_size * sizeof(float));

	if (gpu_error != hipSuccess) {
		std::cout << "Error allocating A" << std::endl;
	}
	gpu_error = hipMalloc((void**)&dev_B, m_size * sizeof(float));
	if (gpu_error != hipSuccess) {
		std::cout << "Error allocating B" << std::endl;
	}
	gpu_error = hipMalloc((void**)&dev_C, m_size * sizeof(float));
	if (gpu_error != hipSuccess) {
		std::cout << "Error allocating C" << std::endl;
	}
	// copy matrices to gpu
	gpu_error = hipMemcpy(dev_A, A, m_size * sizeof(float), hipMemcpyHostToDevice);

	if (gpu_error != hipSuccess) {
		std::cout << "error allocating A video memory: " << hipGetErrorString(gpu_error) << std::endl;
	}
	gpu_error = hipMemcpy(dev_B, B, m_size * sizeof(float), hipMemcpyHostToDevice);

	if (gpu_error != hipSuccess) {
		std::cout << "error allocating B video memory: " << hipGetErrorString(gpu_error) << std::endl;
	}
	hipEvent_t start, stop;
	float gpu_time = 0;
	gpu_error = hipEventCreate(&start);
	if (gpu_error != hipSuccess) {
		std::cout << "Error creating start event: " << hipGetErrorString(gpu_error) << std::endl;
	}
	gpu_error = hipEventCreate(&stop);
	if (gpu_error != hipSuccess) {
		std::cout << "Error creating stop event: " << hipGetErrorString(gpu_error) << std::endl;
	}
	dim3 grid;
	dim3 block;

	int num_blocks = n / B_WIDTH;
	if (n%B_WIDTH) num_blocks++;

	block = dim3(B_WIDTH, B_WIDTH);
	grid = dim3(num_blocks, num_blocks);

	// Add by element
	if (mode == 0) {
		hipEventRecord(start, 0);
		matrixAdd << <grid, block >> >(dev_A, dev_B, dev_C, n);
		gpu_error = hipGetLastError();
		if (gpu_error != hipSuccess) {
			std::cout << "Error during addition by element: " << hipGetErrorString(gpu_error) << std::endl;
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		std::cout << "One thread per element addition (ms): " << gpu_time << std::endl;
	}
	// Add by row
	if (mode == 1) {
		hipEventRecord(start, 0);
		matrixAddRow << <grid, block >> >(dev_A, dev_B, dev_C, n);

		gpu_error = hipGetLastError();
		if (gpu_error != hipSuccess) {
			std::cout << "Error during addition by row: " << hipGetErrorString(gpu_error) << std::endl;
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		std::cout << "One thread per row addition (ms): " << gpu_time << std::endl;
	}
	// Add by column
	if (mode == 2) {
		hipEventRecord(start, 0);
		matrixAddColumn << <grid, block >> >(dev_A, dev_B, dev_C, n);

		gpu_error = hipGetLastError();
		if (gpu_error != hipSuccess) {
			std::cout << "Error during addition by column: " << hipGetErrorString(gpu_error) << std::endl;
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		std::cout << "One thread per column addition (ms): " << gpu_time << std::endl;
	}

	// Add by CPU
	if (mode == 3) {
		float cpu_time = 0;
		hipEventRecord(start, 0);

		for (int i = 0; i < m_size; i++) {
			float t = A[i] + B[i];
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&cpu_time, start, stop);
		std::cout << "CPU add time (ms): " << cpu_time << std::endl;
	}
	gpu_error = hipEventDestroy(start);
	if (gpu_error != hipSuccess) {
		std::cout << "Error destroying start event: " << hipGetErrorString(gpu_error) << std::endl;
	}
	gpu_error = hipEventDestroy(stop);
	if (gpu_error != hipSuccess) {
		std::cout << "Error destroying stop event: " << hipGetErrorString(gpu_error) << std::endl;
	}

	hipDeviceSynchronize();

	gpu_error = hipMemcpy(C, dev_C, m_size * sizeof(float), hipMemcpyDeviceToHost);
	if (gpu_error != hipSuccess) {
		std::cout << "error copying C back to host: " << hipGetErrorString(gpu_error) << std::endl;
	}
	gpu_error = hipFree(dev_A);
	if (gpu_error != hipSuccess) {
		std::cout << "error freeing dev_A: " << hipGetErrorString(gpu_error) << std::endl;
	}
	gpu_error = hipFree(dev_B);
	if (gpu_error != hipSuccess) {
		std::cout << "error freeing dev_B: " << hipGetErrorString(gpu_error) << std::endl;
	}
	hipError_t c_error = hipFree(dev_C);
	if (c_error != hipSuccess) {
		std::cout << "error freeing dev_C: " << hipGetErrorString(c_error) << std::endl;
	}
}

int testAdd(const float* A, const float* B, const float* C, int n) {
	for (int i = 0; i < n*n; i++) {
		if (A[i] + B[i] != C[i]) {
			std::cout << "ERROR AT [" << i / n << ", " << i%n << "]: Incorrect sum. Expected: " << A[i] << " + " << B[i] << " = " << (A[i] + B[i]) << ", Result: " << C[i] << std::endl;
			return 1;
		}
	}
	return 0;
}

void MatrixNAdd(int n) {
	float* A;
	float* B;
	float* C;

	// Initialize 125 x 125 matrix
	int m_size = n*n;
	// init matrices and assign random values
	A = (float*)malloc(m_size * sizeof(float));
	B = (float*)malloc(m_size * sizeof(float));
	C = (float*)malloc(m_size * sizeof(float));

	for (int i = 0; i < m_size; i++) {
		A[i] = rand() % 1000 / 10.0;
		B[i] = rand() % 1000 / 10.0;
	}

	std::cout << "Testing for matrix size: " << n << " x " << n << std::endl;

	// Add by element
	cudaAdd(A, B, C, n, 0);
	int testout = testAdd(A, B, C, n);
	// Add by row
	cudaAdd(A, B, C, n, 1);
	testout += testAdd(A, B, C, n);
	// Add by column
	cudaAdd(A, B, C, n, 2);
	testout += testAdd(A, B, C, n);
	// Add by CPU
	cudaAdd(A, B, C, n, 3);
	testout += testAdd(A, B, C, n);

	if (testout == 0) {
		std::cout << "Test passed!" << std::endl;
	}
	else {
		std::cout << "Test failed!" << std::endl;
	}

	free(A);
	free(B);
	free(C);

}

int main() {
	// init pseudorandom generator so that A and B will not be identical every time
	srand(time(0));
	// 125x125
	MatrixNAdd(125);
	// 250x250
	MatrixNAdd(250);
	// 500x500
	MatrixNAdd(500);	
	//1000x1000
	MatrixNAdd(1000);	
	//2000x2000
	MatrixNAdd(2000);
	return 0;
}